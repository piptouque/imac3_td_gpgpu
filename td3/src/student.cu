#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/
#include <algorithm>
#include "student.hpp"

#include <hip/hip_runtime_api.h>

namespace IMAC
{
    enum
    {
        KERNEL_EX1 = 0,
        KERNEL_EX2,
        KERNEL_EX3,
        KERNEL_EX4,
        KERNEL_EX5
    };

    __device__
    int cuda_getNumberToProcess(const uint arraySize)
    {
        return blockIdx.x == gridDim.x - 1 ? (arraySize - 1) % (2 * blockDim.x) + 1 : 2 * blockDim.x;
    }

    __device__
    void cuda_fillShrArray(const uint* const dev_array, const uint size)
    {
        extern __shared__ uint shr_array[];

        int shr_idx = 2 * threadIdx.x;
        int dev_idx = shr_idx + 2 * blockIdx.x * blockDim.x;
        if (dev_idx < size)
        {
            // printf("shr_idx: %d, dev_idx: %d \n", shr_idx, dev_idx);
            // printf("%d ", dev_idx);
            shr_array[shr_idx] = dev_array[dev_idx];
            if (dev_idx + 1 < size)
            {
                // printf("shr_idx + 1: %d, dev_idx + 1: %d \n", shr_idx + 1, dev_idx + 1);
                // printf("%d ", dev_idx + 1);
                shr_array[shr_idx + 1] = dev_array[dev_idx + 1];
            }
        }
    }

    // ==================================================== EX 1
    __global__
    void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
        extern __shared__ uint shr_array[];
        // account for unused space in case size is not a power of two.
        int numberToProcess = cuda_getNumberToProcess(size);
        // first, copy the input to the shared memory for this block.
        cuda_fillShrArray(dev_array, size);
        // then, in-place max reduction.
        for (int step = 1;; step *= 2)
        {
            int shr_idx = 2 * step * threadIdx.x;
            int shr_next = shr_idx + step;
            if (shr_idx >= numberToProcess || shr_next >= numberToProcess)
            {
                break;
            }
            shr_array[shr_idx] = umax(shr_array[shr_idx], shr_array[shr_next]);
            __syncthreads();
        }
        if (threadIdx.x == 0)
        {
            // lastly, copy from the shared memory to the corresponding space for this block.
            dev_partialMax[blockIdx.x] = shr_array[0];
        }
    }

    // ==================================================== EX 2
    __global__
    void maxReduce_ex2(const uint *const dev_array, const uint size, uint *const dev_partialMax)
    {
        extern __shared__ uint shr_array[];
        // account for unused space in case size is not a power of two.
        int numberToProcess = cuda_getNumberToProcess(size);
        // first, copy the input to the shared memory for this block.
        cuda_fillShrArray(dev_array, size);
        // then, in-place max reduction.
        for (int step = 1;; step *= 2)
        {
            int shr_idx = threadIdx.x;
            int shr_next = shr_idx + (numberToProcess - 1) / (2 * step) + 1;
            if (2 * shr_idx >= shr_next || shr_next >= numberToProcess)
            {
                break;
            }
            shr_array[shr_idx] = umax(shr_array[shr_idx], shr_array[shr_next]);
            __syncthreads();
        }
        if (threadIdx.x == 0)
        {
            // lastly, copy from the shared memory to the corresponding space for this block.
            dev_partialMax[blockIdx.x] = shr_array[0];
        }
    }



    // return a uint2 with x: dimBlock / y: dimGrid
    template<uint kernelType>
    uint2 configureKernel(const uint sizeArray)
    {
        hipDeviceProp_t prop;
        int device;
        HANDLE_ERROR(hipGetDevice(&device));
        HANDLE_ERROR(hipGetDeviceProperties(&prop, device));

        unsigned long maxThreadsPerBlock	= prop.maxThreadsPerBlock;

        uint2 dimBlockGrid; // x: dimBlock / y: dimGrid

        // Configure number of threads/blocks
        switch(kernelType)
        {
            case KERNEL_EX1: case KERNEL_EX2:
                // only allocating a single block of threads if the array length is small.
                dimBlockGrid.x = std::max<uint>(1, std::min<uint>(sizeArray / 2, maxThreadsPerBlock));
                // set number of blocks according to the size of the input array.
                dimBlockGrid.y = std::max<uint>(1, std::max<uint>(1, (sizeArray - 1)) / dimBlockGrid.x);
            case KERNEL_EX3:
                /// TODO EX 3
                break;
            case KERNEL_EX4:
                /// TODO EX 4
                break;
            case KERNEL_EX5:
                /// TODO EX 5
                break;
            default:
                throw std::runtime_error("Error configureKernel: unknown kernel type");
        }
        verifyDimGridBlock( dimBlockGrid.y, dimBlockGrid.x, sizeArray ); // Are you reasonable ?

        return dimBlockGrid;
    }

    // Launch kernel number 'kernelType' and return float2 for timing (x:device,y:host)
    template<uint kernelType>
    float2 reduce(const uint nbIterations, const uint *const dev_array, const uint size, uint &result)
    {
        const uint2 dimBlockGrid = configureKernel<kernelType>(size); // x: dimBlock / y: dimGrid

        // Allocate arrays (host and device) for partial result
        std::vector<uint> host_partialMax(dimBlockGrid.y); // REPLACE SIZE !
        const size_t bytesPartialMax = host_partialMax.size() * sizeof(uint); // REPLACE BYTES !
        const size_t bytesSharedMem = 2 * dimBlockGrid.x * sizeof(uint); // REPLACE BYTES !

        uint *dev_partialMax;
        HANDLE_ERROR(hipMalloc((void**) &dev_partialMax, bytesPartialMax ) );

        std::cout 	<< "Computing on " << dimBlockGrid.y << " block(s) and "
                     << dimBlockGrid.x << " thread(s) "
                     <<"- shared mem size = " << bytesSharedMem << std::endl;

        ChronoGPU chrGPU;
        float2 timing = { 0.f, 0.f }; // x: timing GPU, y: timing CPU
        // Average timing on 'loop' iterations
        for (uint i = 0; i < nbIterations; ++i)
        {
            chrGPU.start();
            switch(kernelType) // Evaluated at compilation time
            {
                case KERNEL_EX1:
                    maxReduce_ex1<<<dimBlockGrid.y, dimBlockGrid.x, bytesSharedMem>>>(dev_array, size, dev_partialMax);
                    break;
                case KERNEL_EX2:
                    maxReduce_ex2<<<dimBlockGrid.y, dimBlockGrid.x, bytesSharedMem>>>(dev_array, size, dev_partialMax);
                    break;
                case KERNEL_EX3:
                    /// TODO EX 3
                    std::cout << "Not implemented !" << std::endl;
                    break;
                case KERNEL_EX4:
                    /// TODO EX 4
                    std::cout << "Not implemented !" << std::endl;
                    break;
                case KERNEL_EX5:
                    /// TODO EX 5
                    std::cout << "Not implemented !" << std::endl;
                    break;
                default:
                    hipFree(dev_partialMax);
                    throw("Error reduce: unknown kernel type.");
            }
            chrGPU.stop();
            timing.x += chrGPU.elapsedTime();
        }
        timing.x /= (float)nbIterations; // Stores time for device

        // Retrieve partial result from device to host
        HANDLE_ERROR(hipMemcpy(host_partialMax.data(), dev_partialMax, bytesPartialMax, hipMemcpyDeviceToHost));

        hipFree(dev_partialMax);

        // Check for error
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }

        ChronoCPU chrCPU;
        chrCPU.start();

        // Finish on host
        for (int i = 0; i < host_partialMax.size(); ++i)
        {
            result = std::max<uint>(result, host_partialMax[i]);
        }
        chrCPU.stop();

        timing.y = chrCPU.elapsedTime(); // Stores time for host

        return timing;
    }
    void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */, const uint nbIterations)
    {
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "Test with " << nbIterations << " iterations" << std::endl;

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(nbIterations, dev_array, array.size(), res1);

        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		std::cout << "========== Ex 2 " << std::endl;
		uint res2 = 0; // result
		// Launch reduction and get timing
		float2 timing2 = reduce<KERNEL_EX2>(nbIterations, dev_array, array.size(), res2);
		
        std::cout << " -> Done: ";
        printTiming(timing2);
		compare(res2, resCPU);

		std::cout << "========== Ex 3 " << std::endl;
		uint res3 = 0; // result
		// Launch reduction and get timing
		float2 timing3 = reduce<KERNEL_EX3>(nbIterations, dev_array, array.size(), res3);
		
        std::cout << " -> Done: ";
        printTiming(timing3);
		compare(res3, resCPU);

		std::cout << "========== Ex 4 " << std::endl;
		uint res4 = 0; // result
		// Launch reduction and get timing
		float2 timing4 = reduce<KERNEL_EX4>(nbIterations, dev_array, array.size(), res4);
		
        std::cout << " -> Done: ";
        printTiming(timing4);
		compare(res4, resCPU);

		std::cout << "========== Ex 5 " << std::endl;
		uint res5 = 0; // result
		// Launch reduction and get timing
		float2 timing5 = reduce<KERNEL_EX5>(nbIterations, dev_array, array.size(), res5);
		
        std::cout << " -> Done: ";
        printTiming(timing5);
		compare(res5, resCPU);

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}
